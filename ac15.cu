#include "hip/hip_runtime.h"
#include<stdio.h>
#include<stdlib.h>
#include<hip/hip_runtime.h>

__global__ void encontraMenor(int *mG_glb,int *menor, int tam)
{
        int idx = threadIdx.x + blockIdx.x*blockDim.x;
        if (idx < tam)
        {
                atomicMin(menor,mG_glb[idx]);
        }
}

FILE *abrir_arquivo(char nome[])
{
        FILE *pFile;
        pFile = fopen(nome, "r");
        if(pFile==NULL)
                exit(0);
        return pFile;
}

int main(int argc,char **argv)
{
        FILE *file;
        int *mA_h,*mB_h;
        int *mA_d,*mB_d;
        int dimA[2],dimB[2];
        int i,j;
        int *menorA_d;
        int *menorA_h;
        int *menorB_d;
        int *menorB_h;
        hipStream_t stream1, stream2;

        hipStreamCreate(&stream1);
        hipStreamCreate(&stream2);

        //abri o arquivo das matrizes
        file = abrir_arquivo(argv[1]);
        //leitura das dimenssões
        fscanf(file,"%d",&dimA[0]);
        fscanf(file,"%d",&dimA[1]);
        fscanf(file,"%d",&dimB[0]);
        fscanf(file,"%d",&dimB[1]);
        //printf("%d %d \n %d %d\n" ,dimA[0], dimA[1], dimB[0], dimB[1]);
        //alocando matrizes e retorno
        //mA_h=(int* )malloc(dimA[0]*dimA[1]*sizeof(int));
        //mB_h=(int* )malloc(dimB[0]*dimB[1]*sizeof(int));
        //menor_h=(int *) malloc(1*sizeof(int));

        hipHostMalloc((void**)&mA_h,(dimA[0])*dimA[1]*(sizeof(int)));
        hipHostMalloc((void**)&mB_h,(dimB[0])*dimB[1]* (sizeof(int)));
        menorA_h=(int *) malloc(1*sizeof(int));
        menorB_h=(int *) malloc(1*sizeof(int));
        //hipHostMalloc((void**)&menorA_h,sizeof(int));
        //hipHostMalloc((void**)&menorB_h,sizeof(int));



        //Leitura do arquivo para matriz A e B
        for(i=0;i<dimA[0];i++)
                for(j=0;j<dimA[1];j++)
                        fscanf(file,"%d", &mA_h[i*dimA[1]+j]);

        for(i=0;i<dimB[0];i++)
                for(j=0;j<dimB[1];j++)
                        fscanf(file,"%d", &mB_h[i*dimB[1]+j]);
        fclose(file);

        hipMalloc((void**)&mA_d,(dimA[0])*dimA[1]*(sizeof(int)));
        hipMalloc((void**)&mB_d,(dimB[0])*dimB[1]* (sizeof(int)));
        hipMalloc((void**)&menorA_d,sizeof(int));
        hipMalloc((void**)&menorB_d,sizeof(int));

        //Inicializa o conteúdo da variável no device com 10000
        hipMemset(menorA_d,10000,sizeof(int));
        hipMemset(menorB_d,10000,sizeof(int));
        
        int threadsPerBlock = 2;
        int blocksPerGridA = ((dimA[0])+threadsPerBlock-1)/threadsPerBlock;
        int blocksPerGridB = ((dimB[0])+threadsPerBlock-1)/threadsPerBlock;

        //copia arquivos do host para o device
        hipMemcpyAsync(mA_d,mA_h,(dimA[0]*dimA[1]*sizeof(int)), hipMemcpyHostToDevice, stream1);
        hipMemcpyAsync(mB_d,mB_h,(dimB[0]*dimA[1]*sizeof(int)), hipMemcpyHostToDevice, stream2);

        encontraMenor <<<blocksPerGridA,threadsPerBlock,0,stream1>>>(mA_d,&menorA_d,dimA[0]*dimA[1]);
        hipMemcpy(menorA_h,menorA_d,sizeof(int), hipMemcpyDeviceToHost);

        encontraMenor <<<blocksPerGridB,threadsPerBlock,0,stream2>>>(mB_d,&menorB_d,dimB[0]*dimB[1]);
        hipMemcpy(menorB_h,menorB_d,sizeof(int), hipMemcpyDeviceToHost);

        hipStreamSynchronize(stream1);
        hipStreamSynchronize(stream2);

        printf("Menor valor da matriz A: %d\n", menorA_h);
        printf("Menor valor da matriz B: %d\n", menorB_h);

        printf("Matriz A - Flatten\n");
        for(i=0;i<dimA[0];i++)
                for(j=0;j<dimA[1];j++)
                        printf("%d ", mA_h[i*dimA[1]+j]);
                printf("\n\n");
        printf("Matriz B - Flatten\n");
        for(i=0;i<dimB[0];i++)
                for(j=0;j<dimB[1];j++)
                        printf("%d ", mB_h[i*dimB[1]+j]);
        printf("\n");

        hipHostFree(menorA_h);
        hipHostFree(menorB_h);
        hipHostFree(mA_h);
        hipHostFree(mB_h);

        hipFree(mB_d);
        hipFree(mA_d);
        hipFree(menorA_d);
        hipFree(menorB_d);

        hipStreamDestroy(stream1);
        hipStreamDestroy(stream2);

        //chama a função com o stream
        //encontraMenor <<<BPG,TPB>>> (mA_d,menor_d,dimA[0]*dimA[1]);
        //
        //Copia o resultado da soma de volta para o host
        //hipMemcpy(menor_h,menor_d,sizeof(int), hipMemcpyDeviceToHost);
        //sincroniza os valores
        //hipStreamSynchronize(stream);
        //printf("Menor elemento da matriz A\n");
        //printf("%d\n\n",*menor_h);
        //imprime
        //free(menor_h);
        //free(mA_h);
        //free(mB_h);
        //hipFree(menor_d);
        //Desaloca os vetores no device
        //hipFree(mB_d);
        //hipFree(mA_d);
        exit(0);
}