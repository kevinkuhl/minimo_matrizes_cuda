#include<stdio.h>
#include<stdlib.h>
#include<hip/hip_runtime.h>

__global__ void encontraMenor(int *mG_glb,int *menor, int tam)
{
        int idx = threadIdx.x + blockIdx.x*blockDim.x;
        if (idx < tam)
        {
                atomicMin(menor,mG_glb[idx]);
        }
}

FILE *abrir_arquivo(char nome[])
{
        FILE *pFile;
        pFile = fopen(nome, "r");
        if(pFile==NULL)
                exit(0);
        return pFile;
}

int main(int argc,char **argv)
{
        FILE *file;
        int *mA_h,*mB_h;
        int *mA_d,*mB_d;
        int dimA[2],dimB[2];
        int i,j;
        int *menorA_d;
        int *menorA_h;
        int *menorB_d;
        int *menorB_h;
        hipStream_t stream1, stream2;

        hipStreamCreate(&stream1);
        hipStreamCreate(&stream2);

        //abri o arquivo das matrizes
        file = abrir_arquivo(argv[1]);
        //leitura das dimenssões
        fscanf(file,"%d",&dimA[0]);
        fscanf(file,"%d",&dimA[1]);
        fscanf(file,"%d",&dimB[0]);
        fscanf(file,"%d",&dimB[1]);

        hipHostMalloc((void**)&mA_h,(dimA[0])*dimA[1]*(sizeof(int)), hipHostMallocDefault);
        hipHostMalloc((void**)&mB_h,(dimB[0])*dimB[1]* (sizeof(int)), hipHostMallocDefault);
        // menorA_h=(int *) malloc(1*sizeof(int));
        // menorB_h=(int *) malloc(1*sizeof(int));
        hipHostMalloc((void**)&menorA_h,sizeof(int), hipHostMallocDefault);
        hipHostMalloc((void**)&menorB_h,sizeof(int), hipHostMallocDefault);



        //Leitura do arquivo para matriz A e B
        for(i=0;i<dimA[0];i++)
                for(j=0;j<dimA[1];j++)
                        fscanf(file,"%d", &mA_h[i*dimA[1]+j]);

        for(i=0;i<dimB[0];i++)
                for(j=0;j<dimB[1];j++)
                        fscanf(file,"%d", &mB_h[i*dimB[1]+j]);
        fclose(file);

        hipMalloc((void**)&mA_d,(dimA[0])*dimA[1]*(sizeof(int)));
        hipMalloc((void**)&mB_d,(dimB[0])*dimB[1]* (sizeof(int)));
        hipMalloc((void**)&menorA_d,sizeof(int));
        hipMalloc((void**)&menorB_d,sizeof(int));

        //Inicializa o conteúdo da variável no device com 10000
        hipMemset(menorA_d,10000,sizeof(int));
        hipMemset(menorB_d,10000,sizeof(int));
        
        int threadsPerBlockA = dimA[0]*dimA[1];
        int threadsPerBlockB = dimB[0]*dimB[1];
        int blocksPerGridA = ((dimA[0])+threadsPerBlockA-1)/threadsPerBlockA;
        int blocksPerGridB = ((dimB[0])+threadsPerBlockB-1)/threadsPerBlockB;

        //copia arquivos do host para o device
        hipMemcpyAsync(mA_d,mA_h,(dimA[0]*dimA[1]*sizeof(int)), hipMemcpyHostToDevice, stream1);
        hipMemcpyAsync(mB_d,mB_h,(dimB[0]*dimB[1]*sizeof(int)), hipMemcpyHostToDevice, stream2);

        encontraMenor <<<blocksPerGridA,threadsPerBlockA,0,stream1>>>(mA_d,menorA_d,dimA[0]*dimA[1]);
        hipMemcpy(menorA_h,menorA_d,sizeof(int), hipMemcpyDeviceToHost);

        encontraMenor <<<blocksPerGridB,threadsPerBlockB,0,stream2>>>(mB_d,menorB_d,dimB[0]*dimB[1]);
        hipMemcpy(menorB_h,menorB_d,sizeof(int), hipMemcpyDeviceToHost);

        hipStreamSynchronize(stream1);
        hipStreamSynchronize(stream2);

        printf("Menor valor da matriz A: %d\n", *menorA_h);
        printf("Menor valor da matriz B: %d\n", *menorB_h);

        printf("Matriz A - Flatten\n");
        for(i=0;i<dimA[0];i++)
                for(j=0;j<dimA[1];j++)
                        printf("%d ", mA_h[i*dimA[1]+j]);
                printf("\n\n");

        printf("Matriz B - Flatten\n");
        for(i=0;i<dimB[0];i++)
                for(j=0;j<dimB[1];j++)
                        printf("%d ", mB_h[i*dimB[1]+j]);
        printf("\n");

        hipHostFree(menorA_h);
        hipHostFree(menorB_h);
        hipHostFree(mA_h);
        hipHostFree(mB_h);

        hipFree(mB_d);
        hipFree(mA_d);
        hipFree(menorA_d);
        hipFree(menorB_d);

        hipStreamDestroy(stream1);
        hipStreamDestroy(stream2);
        
        exit(0);
}